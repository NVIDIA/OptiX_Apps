#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.h"

#include <optix.h>

#include "system_parameter.h"
#include "per_ray_data.h"
#include "vertex_attributes.h"
#include "material_parameter.h"
#include "function_indices.h"
#include "light_definition.h"
#include "shader_common.h"
#include "random_number_generators.h"


extern "C" __constant__ SystemParameter sysParameter;


// Get the 3x4 object to world transform and its inverse from a two-level hierarchy.
// Arguments float4* objectToWorld, float4* worldToObject shortened for smaller code.
__forceinline__ __device__ void getTransforms(float4* mW, float4* mO) 
{
  OptixTraversableHandle handle = optixGetTransformListHandle(0);
  
  const float4* tW = optixGetInstanceTransformFromHandle(handle);
  const float4* tO = optixGetInstanceInverseTransformFromHandle(handle);

  mW[0] = tW[0];
  mW[1] = tW[1];
  mW[2] = tW[2];

  mO[0] = tO[0];
  mO[1] = tO[1];
  mO[2] = tO[2];
}

// Functions to get the individual transforms in case only one of them is needed.

__forceinline__ __device__ void getTransformObjectToWorld(float4* mW) 
{
  OptixTraversableHandle handle = optixGetTransformListHandle(0);
  
  const float4* tW = optixGetInstanceTransformFromHandle(handle);

  mW[0] = tW[0];
  mW[1] = tW[1];
  mW[2] = tW[2];
}

__forceinline__ __device__ void getTransformWorldToObject(float4* mO) 
{
  OptixTraversableHandle handle = optixGetTransformListHandle(0);
  
  const float4* tO = optixGetInstanceInverseTransformFromHandle(handle);

  mO[0] = tO[0];
  mO[1] = tO[1];
  mO[2] = tO[2];
}


// Matrix3x4 * point. v.w == 1.0f
__forceinline__ __device__ float3 transformPoint(const float4* m, float3 const& v)
{
  float3 r;

  r.x = m[0].x * v.x + m[0].y * v.y + m[0].z * v.z + m[0].w;
  r.y = m[1].x * v.x + m[1].y * v.y + m[1].z * v.z + m[1].w;
  r.z = m[2].x * v.x + m[2].y * v.y + m[2].z * v.z + m[2].w;

  return r;
}

// Matrix3x4 * vector. v.w == 0.0f
__forceinline__ __device__ float3 transformVector(const float4* m, float3 const& v)
{
  float3 r;

  r.x = m[0].x * v.x + m[0].y * v.y + m[0].z * v.z;
  r.y = m[1].x * v.x + m[1].y * v.y + m[1].z * v.z;
  r.z = m[2].x * v.x + m[2].y * v.y + m[2].z * v.z;

  return r;
}

// InverseMatrix3x4^T * normal. v.w == 0.0f
// Get the inverse matrix as input and applies it as inverse transpose.
__forceinline__ __device__ float3 transformNormal(const float4* m, float3 const& v)
{
  float3 r;

  r.x = m[0].x * v.x + m[1].x * v.y + m[2].x * v.z;
  r.y = m[0].y * v.x + m[1].y * v.y + m[2].y * v.z;
  r.z = m[0].z * v.x + m[1].z * v.y + m[2].z * v.z;

  return r;
}


extern "C" __global__ void __closesthit__radiance()
{
  GeometryInstanceData* theData = reinterpret_cast<GeometryInstanceData*>(optixGetSbtDataPointer());

  const unsigned int thePrimtiveIndex = optixGetPrimitiveIndex();

  const int3 tri = theData->indices[thePrimtiveIndex];

  const VertexAttributes& va0 = theData->attributes[tri.x];
  const VertexAttributes& va1 = theData->attributes[tri.y];
  const VertexAttributes& va2 = theData->attributes[tri.z];

  const float2 theBarycentrics = optixGetTriangleBarycentrics(); // beta and gamma
  const float  alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;

  const float3 ng = cross(va1.vertex - va0.vertex, va2.vertex - va0.vertex);
  //const float3 tg = va0.tangent * alpha + va1.tangent * theBarycentrics.x + va2.tangent * theBarycentrics.y;
  const float3 ns = va0.normal  * alpha + va1.normal  * theBarycentrics.x + va2.normal  * theBarycentrics.y;
  
  State state; // All in world space coordinates!

  state.texcoord = va0.texcoord * alpha + va1.texcoord * theBarycentrics.x + va2.texcoord * theBarycentrics.y;

  //float4 objectToWorld[3];
  float4 worldToObject[3];
  
  //getTransforms(objectToWorld, worldToObject);

  //getTransformObjectToWorld(objectToWorld);
  getTransformWorldToObject(worldToObject);

  state.normalGeo = normalize(transformNormal(worldToObject, ng));
  //state.tangent   = normalize(transformVector(objectToWorld, tg));
  state.normal    = normalize(transformNormal(worldToObject, ns));

  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  thePrd->distance = optixGetRayTmax(); // Return the current path segment distance, needed for absorption calculations in the integrator.
  
  //thePrd->pos = optixGetWorldRayOrigin() + optixGetWorldRayDirection() * optixGetRayTmax();
  thePrd->pos = thePrd->pos + thePrd->wi * thePrd->distance; // DEBUG Check which version is more efficient.

  // Explicitly include edge-on cases as frontface condition!
  // Keeps the material stack from overflowing at silhouettes.
  // Prevents that silhouettes of thin-walled materials use the backface material.
  // Using the true geometry normal attribute as originally defined on the frontface!
  thePrd->flags |= (0.0f <= dot(thePrd->wo, state.normalGeo)) ? FLAG_FRONTFACE : 0;

  if ((thePrd->flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
  {
    // Means geometric normal and shading normal are always defined on the side currently looked at.
    // This gives the backfaces of opaque BSDFs a defined result.
    state.normalGeo = -state.normalGeo;
    //state.tangent   = -state.tangent;
    state.normal    = -state.normal;
    // Explicitly DO NOT recalculate the frontface condition!
  }
  
  thePrd->radiance = make_float3(0.0f);

  // When hitting a geometric light, evaluate the emission first, because this needs the previous diffuse hit's pdf.
  if (0 <= theData->lightIndex &&       // This material is emissive and
      (thePrd->flags & FLAG_FRONTFACE)) // we're looking at the front face.
  {
    const float cosTheta = dot(thePrd->wo, state.normalGeo);
    if (DENOMINATOR_EPSILON < cosTheta)
    {
      LightDefinition const& light = sysParameter.lightDefinitions[theData->lightIndex];

      float3 emission = light.emission;

#if USE_NEXT_EVENT_ESTIMATION
      const float lightPdf = (thePrd->distance * thePrd->distance) / (light.area * cosTheta); // This assumes the light.area is greater than zero.

      // If it's an implicit light hit from a diffuse scattering event and the light emission was not returning a zero pdf (e.g. backface or edge on).
      if ((thePrd->flags & FLAG_DIFFUSE) && DENOMINATOR_EPSILON < lightPdf)
      {
        // Scale the emission with the power heuristic between the initial BSDF sample pdf and this implicit light sample pdf.
        emission *= powerHeuristic(thePrd->pdf, lightPdf);
      }
#endif // USE_NEXT_EVENT_ESTIMATION

      thePrd->radiance = emission;
      
      // PERF End the path when hitting a light. Emissive materials with a non-black BSDF would normally just continue.
      thePrd->flags |= FLAG_TERMINATE;
      return;
    }
  }

  // Start fresh with the next BSDF sample. (Either of these values remaining zero is an end-of-path condition.)
  // The pdf of the previous evene was needed for the emission calculation above.
  thePrd->f_over_pdf = make_float3(0.0f);
  thePrd->pdf        = 0.0f;

  MaterialParameter const& parameters = sysParameter.materialParameters[theData->materialIndex]; // Use a const reference, not all BSDFs need all values.

  state.albedo = parameters.albedo; // PERF Copy only this locally to be able to modulate it with the optional texture.

  if (parameters.textureAlbedo != 0)
  {
    const float3 texColor = make_float3(tex2D<float4>(parameters.textureAlbedo, state.texcoord.x, state.texcoord.y));

    // Modulate the incoming color with the texture.
    state.albedo *= texColor;               // linear color, resp. if the texture has been uint8 and readmode set to use sRGB, then sRGB.
    //state.albedo *= powf(texColor, 2.2f); // sRGB gamma correction done manually.
  }

  // Only the last diffuse hit is tracked for multiple importance sampling of implicit light hits.
  thePrd->flags = (thePrd->flags & ~FLAG_DIFFUSE) | parameters.flags; // FLAG_THINWALLED can be set directly from the material parameters.

  const int indexBSDF = NUM_LENS_SHADERS + NUM_LIGHT_TYPES + parameters.indexBSDF * 2;

  optixDirectCall<void, MaterialParameter const&, State const&, PerRayData*>(indexBSDF, parameters, state, thePrd);

#if USE_NEXT_EVENT_ESTIMATION
  // Direct lighting if the sampled BSDF was diffuse and any light is in the scene.
  const int numLights = sysParameter.numLights;
  if ((thePrd->flags & FLAG_DIFFUSE) && 0 < numLights)
  {
    // Sample one of many lights. 
    const float2 sample = rng2(thePrd->seed); // Use lower dimension samples for the position. (Irrelevant for the LCG).

    // The caller picks the light to sample. Make sure the index stays in the bounds of the sysParameter.lightDefinitions array.
    const int indexLight = (1 < numLights) ? clamp(static_cast<int>(floorf(rng(thePrd->seed) * numLights)), 0, numLights - 1) : 0;
    
    LightDefinition const& light = sysParameter.lightDefinitions[indexLight];
    
    const int indexCallable = NUM_LENS_SHADERS + light.type;

    LightSample lightSample = optixDirectCall<LightSample, LightDefinition const&, const float3, const float2>(indexCallable, light, thePrd->pos, sample);

    if (0.0f < lightSample.pdf) // Useful light sample?
    {
      // Evaluate the BSDF in the light sample direction. Normally cheaper than shooting rays.
      // Returns BSDF f in .xyz and the BSDF pdf in .w
      // BSDF eval function is one index after the sample fucntion.
      const float4 bsdf_pdf = optixDirectCall<float4, MaterialParameter const&, State const&, PerRayData const*, const float3>(indexBSDF + 1, parameters, state, thePrd, lightSample.direction);

      if (0.0f < bsdf_pdf.w && isNotNull(make_float3(bsdf_pdf)))
      {
        // Pass the current payload registers through to the shadow ray.
        unsigned int p0 = optixGetPayload_0();
        unsigned int p1 = optixGetPayload_1();

        // Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval 
        // to prevent self-intersections with the actual light geometry in the scene.
        optixTrace(sysParameter.topObject,
                   thePrd->pos, lightSample.direction, // origin, direction
                   sysParameter.sceneEpsilon, lightSample.distance - sysParameter.sceneEpsilon, 0.0f, // tmin, tmax, time
                   OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT, // The shadow ray type only uses anyhit programs.
                   RAYTYPE_SHADOW, NUM_RAYTYPES, RAYTYPE_SHADOW,
                   p0, p1); // Pass through thePrd to the shadow ray. It needs the seed and sets flags.

        if ((thePrd->flags & FLAG_SHADOW) == 0) // Shadow flag not set?
        {
          if (thePrd->flags & FLAG_VOLUME) // Supporting nested materials includes having lights inside a volume.
          {
            // Calculate the transmittance along the light sample's distance in case it's inside a volume.
            // The light must be in the same volume or it would have been shadowed!
            lightSample.emission *= expf(-lightSample.distance * thePrd->extinction);
          }

          const float misWeight = powerHeuristic(lightSample.pdf, bsdf_pdf.w);
            
          thePrd->radiance += make_float3(bsdf_pdf) * lightSample.emission * (misWeight * dot(lightSample.direction, state.normal) / lightSample.pdf);
        }
      }
    }
  }
#endif // USE_NEXT_EVENT_ESTIMATION
}
