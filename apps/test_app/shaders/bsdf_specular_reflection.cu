#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.h"

#include <optix.h>

#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"

extern "C" __device__ void __direct_callable__sample_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData* prd)
{
  prd->wi = reflect(-prd->wo, state.normal);

  if (dot(prd->wi, state.normalGeo) <= 0.0f) // Do not sample opaque materials below the geometric surface.
  {
    prd->flags |= FLAG_TERMINATE;
    return;
  }

  prd->f_over_pdf = state.albedo;
  prd->pdf        = 1.0f; // Not 0.0f to make sure the path is not terminated. Otherwise unused for specular events.
}

// This is actually never reached, because the FLAG_DIFFUSE flag is not set when a specular BSDF is has been sampled.
extern "C" __device__ float4 __direct_callable__eval_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData* const prd, const float3 wiL)
{
  return make_float4(0.0f);
}
