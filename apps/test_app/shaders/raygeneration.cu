#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.h"

#include <optix.h>

#include "system_parameter.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "random_number_generators.h"


extern "C" __constant__ SystemParameter sysParameter;

extern "C" __global__ void __raygen__pathtracer()
{
  PerRayData prd;

  // This assumes that the launch dimensions are matching the size of the output buffer.
  const uint3 theLaunchDim   = optixGetLaunchDimensions();
  const uint3 theLaunchIndex = optixGetLaunchIndex();

  // Initialize the random number generator seed from the linear pixel index and the iteration index.
  prd.seed = tea<4>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, sysParameter.iterationIndex);

  // Decoupling the pixel coordinates from the screen size will allow for partial rendering algorithms.
  // In this case theLaunchIndex is the pixel coordinate and theLaunchDim is sysOutputBuffer.size().
  const float2 screen = make_float2(theLaunchDim);
  const float2 pixel  = make_float2(theLaunchIndex);
  const float2 sample = rng2(prd.seed);

  // Lens shaders
  optixDirectCall<void, const float2, const float2, const float2, float3&, float3&>(sysParameter.cameraType, screen, pixel, sample, prd.pos, prd.wi);

  // This renderer supports nested volumes. Four levels is plenty enough for most cases.
  // The absorption coefficient and IOR of the volume the ray is currently inside.
  float4 absorptionStack[MATERIAL_STACK_SIZE]; // .xyz == absorptionCoefficient (sigma_a), .w == index of refraction
  
  float3 radiance   = make_float3(0.0f); // Start with black.
  float3 throughput = make_float3(1.0f); // The throughput for the next radiance, starts with 1.0f.

  int stackIdx = MATERIAL_STACK_EMPTY; // Start with empty nested materials stack.

  // Russian Roulette path termination after a specified number of bounces needs the current depth.
  int depth = 0; // Path segment index. Primary ray is 0. 

  prd.absorption_ior = make_float4(0.0f, 0.0f, 0.0f, 1.0f); // Assume primary ray starts in vacuum.
  prd.flags          = 0;

  while (depth < sysParameter.pathLengths.y)
  {
    prd.wo        = -prd.wi;            // Direction to observer.
    prd.ior       = make_float2(1.0f);  // Reset the volume IORs.
    prd.distance  = RT_DEFAULT_MAX;     // Shoot the next ray with maximum length.
    prd.flags    &= FLAG_CLEAR_MASK;    // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.

    // Handle volume absorption of nested materials.
    if (MATERIAL_STACK_FIRST <= stackIdx) // Inside a volume?
    {
      prd.flags     |= FLAG_VOLUME;                            // Indicate that we're inside a volume. => At least absorption calculation needs to happen.
      prd.extinction = make_float3(absorptionStack[stackIdx]); // There is only volume absorption in this demo, no volume scattering.
      prd.ior.x      = absorptionStack[stackIdx].w;            // The IOR of the volume we're inside. Needed for eta calculations in transparent materials.
      if (MATERIAL_STACK_FIRST <= stackIdx - 1)
      {
        prd.ior.y = absorptionStack[stackIdx - 1].w; // The IOR of the surrounding volume. Needed when potentially leaving a volume to calculate eta in transparent materials.
      }
    }

    // Note that the primary rays (or volume scattering miss cases) wouldn't normally offset the ray t_min by sysSceneEpsilon. Keep it simple here.

    // Put radiance payload pointer into two unsigned integers.
    uint2 payload = splitPointer(&prd);

    optixTrace(sysParameter.topObject,
               prd.pos, prd.wi, // origin, direction
               sysParameter.sceneEpsilon, prd.distance, 0.0f, // tmin, tmax, time
               OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_NONE, 
               RAYTYPE_RADIANCE, NUM_RAYTYPES, RAYTYPE_RADIANCE,
               payload.x, payload.y);

    // This renderer supports nested volumes.
    if (prd.flags & FLAG_VOLUME)
    {
      // We're inside a volume. Calculate the extinction along the current path segment in any case.
      // The transmittance along the current path segment inside a volume needs to attenuate the ray throughput with the extinction
      // before it modulates the radiance of the hitpoint.
      throughput *= expf(-prd.distance * prd.extinction);
    }

    radiance += throughput * prd.radiance;

    // Path termination by miss shader or sample() routines.
    // If terminate is true, f_over_pdf and pdf might be undefined.
    if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
    {
      break;
    }

    // PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
    throughput *= prd.f_over_pdf;

    // Unbiased Russian Roulette path termination.
    if (sysParameter.pathLengths.x <= depth) // Start termination after a minimum number of bounces.
    {
      const float probability = fmaxf(throughput); // Other options: // intensity(throughput); // fminf(0.5f, intensity(throughput));
      if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
      {
        break;
      }
      throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
    }

    // Adjust the material volume stack if the geometry is not thin-walled but a border between two volumes 
    // and the outgoing ray direction was a transmission.
    if ((prd.flags & (FLAG_THINWALLED | FLAG_TRANSMISSION)) == FLAG_TRANSMISSION) 
    {
      // Transmission.
      if (prd.flags & FLAG_FRONTFACE) // Entered a new volume?
      {
        // Push the entered material's volume properties onto the volume stack.
        //rtAssert((stackIdx < MATERIAL_STACK_LAST), 1); // Overflow?
        stackIdx = min(stackIdx + 1, MATERIAL_STACK_LAST);
        absorptionStack[stackIdx] = prd.absorption_ior;
      }
      else // Exited the current volume?
      {
        // Pop the top of stack material volume.
        // This assert fires and is intended because I tuned the frontface checks so that there are more exits than enters at silhouettes.
        //rtAssert((MATERIAL_STACK_EMPTY < stackIdx), 0); // Underflow?
        stackIdx = max(stackIdx - 1, MATERIAL_STACK_EMPTY);
      }
    }

    ++depth; // Next path segment.
  }

#if USE_DEBUG_EXCEPTIONS
  // DEBUG Highlight numerical errors.
  if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
  {
    radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
  }
  else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
  {
    radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
  }
  else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
  {
    radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
  }
#else
  // NaN values will never go away. Filter them out before they can arrive in the output buffer.
  // This only has an effect if the debug coloring above is off!
  if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
  {
    const unsigned int index = theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x;
    if (0 < sysParameter.iterationIndex)
    {
      const float4 dst = sysParameter.outputBuffer[index]; // RGBA32F
      radiance = lerp(make_float3(dst), radiance, 1.0f / float(sysParameter.iterationIndex + 1));
    }
    // sysIterationIndex 0 will fill the buffer.
    // If this isn't done separately, the result of the lerp() above is undefined, e.g. dst could be NaN.
    sysParameter.outputBuffer[index] = make_float4(radiance, 1.0f);
  }
}
