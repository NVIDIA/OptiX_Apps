#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"

#include "shader_common.h"

extern "C" __constant__ SystemData sysData;


__forceinline__ __device__ void unitSquareToSphere(const float u, const float v, float3& p, float& pdf)
{
  p.z = 1.0f - 2.0f * u;
  float r = 1.0f - p.z * p.z;
  r = (0.0f < r) ? sqrtf(r) : 0.0f;
  
  const float phi = v * 2.0f * M_PIf;
  p.x = r * cosf(phi);
  p.y = r * sinf(phi);

  pdf = 0.25f * M_1_PIf;  // == 1.0f / (4.0f * M_PIf)
}

// Note that all light sampling routines return lightSample.direction and lightSample.distance in world space!

extern "C" __device__ LightSample __direct_callable__light_env_constant(const LightDefinition& light, const float3 point, const float2 sample)
{
  LightSample lightSample;

  unitSquareToSphere(sample.x, sample.y, lightSample.direction, lightSample.pdf);

  // Environment lights do not set the light sample position!
  lightSample.distance = RT_DEFAULT_MAX; // Environment light.
  
  // Explicit light sample. White scaled by inverse probabilty to hit this light.
  // FIXME Could use the sysData.lightDefinitions[0].emission for different colors.
  lightSample.emission = make_float3(sysData.numLights);
  
  return lightSample;
}

extern "C" __device__ LightSample __direct_callable__light_env_sphere(const LightDefinition& light, const float3 point, const float2 sample)
{
  LightSample lightSample;

  // Importance-sample the spherical environment light direction.
  
  // Note that the marginal CDF is one bigger than the texture height. As index this is the 1.0f at the end of the CDF.
  const unsigned int sizeV = sysData.envHeight;

  unsigned int ilo = 0;     // Use this for full spherical lighting. (This matches the result of indirect environment lighting.)
  unsigned int ihi = sizeV; // Index on the last entry containing 1.0f. Can never be reached with the sample in the range [0.0f, 1.0f).

  const float* cdfV = sysData.envCDF_V;

  // Binary search the row index to look up.
  while (ilo != ihi - 1) // When a pair of limits have been found, the lower index indicates the cell to use.
  {
    const unsigned int i = (ilo + ihi) >> 1;
    if (sample.y < cdfV[i]) // If the cdf is greater than the sample, use that as new higher limit.
    {
      ihi = i;
    }
    else // If the sample is greater than or equal to the CDF value, use that as new lower limit.
    {
      ilo = i; 
    }
  }

  const unsigned int vIdx = ilo; // This is the row we found.
    
  // Note that the horizontal CDF is one bigger than the texture width. As index this is the 1.0f at the end of the CDF.
  const unsigned int sizeU = sysData.envWidth; // Note that the horizontal CDFs are one bigger than the texture width.

  // Binary search the column index to look up.
  ilo = 0;
  ihi = sizeU; // Index on the last entry containing 1.0f. Can never be reached with the sample in the range [0.0f, 1.0f).

  // Pointer to the indexY row!
  const float* cdfU = &sysData.envCDF_U[vIdx * (sizeU + 1)]; // Horizontal CDF is one bigger then the texture width!

  while (ilo != ihi - 1) // When a pair of limits have been found, the lower index indicates the cell to use.
  {
    const unsigned int i = (ilo + ihi) >> 1;
    if (sample.x < cdfU[i]) // If the CDF value is greater than the sample, use that as new higher limit.
    {
      ihi = i;
    }
    else // If the sample is greater than or equal to the CDF value, use that as new lower limit.
    {
      ilo = i;
    }
  }

  const unsigned int uIdx = ilo; // The column result.

  // Continuous sampling of the CDF.
  const float cdfLowerU = cdfU[uIdx];
  const float cdfUpperU = cdfU[uIdx + 1];
  const float du = (sample.x - cdfLowerU) / (cdfUpperU - cdfLowerU);

  const float cdfLowerV = cdfV[vIdx];
  const float cdfUpperV = cdfV[vIdx + 1];
  const float dv = (sample.y - cdfLowerV) / (cdfUpperV - cdfLowerV);

  // Texture lookup coordinates.
  const float u = (float(uIdx) + du) / float(sizeU);
  const float v = (float(vIdx) + dv) / float(sizeV);

  // Light sample direction vector polar coordinates. This is where the environment rotation happens!
  // DAR FIXME Use a light.matrix to rotate the resulting vector instead.
  const float phi   = (u - sysData.envRotation) * 2.0f * M_PIf;
  const float theta = v * M_PIf; // theta == 0.0f is south pole, theta == M_PIf is north pole.

  const float sinTheta = sinf(theta);
  // The miss program places the 1->0 seam at the positive z-axis and looks from the inside.
  lightSample.direction = make_float3(-sinf(phi) * sinTheta,  // Starting on positive z-axis going around clockwise (to negative x-axis).
                                      -cosf(theta),           // From south pole to north pole.
                                       cosf(phi) * sinTheta); // Starting on positive z-axis.

  // Note that environment lights do not set the light sample position!
  lightSample.distance = RT_DEFAULT_MAX; // Environment light.

  const float3 emission = make_float3(tex2D<float4>(sysData.envTexture, u, v));
  // Explicit light sample. The returned emission must be scaled by the inverse probability to select this light.
  lightSample.emission = emission * sysData.numLights;
  // For simplicity we pretend that we perfectly importance-sampled the actual texture-filtered environment map
  // and not the Gaussian-smoothed one used to actually generate the CDFs and uniform sampling in the texel.
  lightSample.pdf = intensity(emission) / sysData.envIntegral;

  return lightSample;
}

extern "C" __device__ LightSample __direct_callable__light_parallelogram(const LightDefinition& light, const float3 point, const float2 sample)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)

  lightSample.position  = light.position + light.vecU * sample.x + light.vecV * sample.y; // The light sample position in world coordinates.
  lightSample.direction = lightSample.position - point; // Sample direction from surface point to light sample position.
  lightSample.distance  = length(lightSample.direction);
  if (DENOMINATOR_EPSILON < lightSample.distance)
  {
    lightSample.direction /= lightSample.distance; // Normalized direction to light.
 
    const float cosTheta = dot(-lightSample.direction, light.normal);
    if (DENOMINATOR_EPSILON < cosTheta) // Only emit light on the front side.
    {
      // Explicit light sample, must scale the emission by inverse probabilty to hit this light.
      lightSample.emission = light.emission * float(sysData.numLights); 
      lightSample.pdf      = (lightSample.distance * lightSample.distance) / (light.area * cosTheta); // Solid angle pdf. Assumes light.area != 0.0f.
    }
  }

  return lightSample;
}
