#include "hip/hip_runtime.h"
 /* 
 * Copyright (c) 2013-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"

#include "per_ray_data.h"
#include "random_number_generators.h"
#include "shader_common.h"
#include "transform.h"

extern "C" __constant__ SystemData sysData;


// Note that all light sampling routines return lightSample.direction and lightSample.distance in world space!

extern "C" __device__ LightSample __direct_callable__light_env_constant(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  const float2 sample = rng2(prd->seed);

  unitSquareToSphere(sample.x, sample.y, lightSample.direction, lightSample.pdf);
  
  // The emission is constant in all directions.
  // There is no transformation of the object space direction into world space necessary.

  lightSample.distance = RT_DEFAULT_MAX; // Environment light.
  
  lightSample.radiance_over_pdf = light.emission / lightSample.pdf;

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_env_sphere(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f;

  // Importance-sample the spherical environment light direction in object space.
  // FIXME The binary searches are generating a lot of memory traffic. Replace this with an alias-map lookup.
  const float2 sample = rng2(prd->seed);

  // Note that the marginal CDF is one bigger than the texture height. As index this is the 1.0f at the end of the CDF.
  const float* cdfV = reinterpret_cast<const float*>(light.cdfV);
  const unsigned int idxV = binarySearchCDF(cdfV, light.height, sample.y);

  const float* cdfU = reinterpret_cast<const float*>(light.cdfU);
  cdfU += (light.width + 1) * idxV; // Horizontal CDF is one bigger than the texture width!
  const unsigned int idxU = binarySearchCDF(cdfU, light.width, sample.x);

  // Continuous sampling of the CDF.
  const float cdfLowerU = cdfU[idxU];
  const float cdfUpperU = cdfU[idxU + 1];
  const float du = (sample.x - cdfLowerU) / (cdfUpperU - cdfLowerU);
  const float u = (float(idxU) + du) / float(light.width);

  const float cdfLowerV = cdfV[idxV];
  const float cdfUpperV = cdfV[idxV + 1];
  const float dv = (sample.y - cdfLowerV) / (cdfUpperV - cdfLowerV);
  const float v = (float(idxV) + dv) / float(light.height);

  // Light sample direction vector in object space polar coordinates.
  const float phi   = u * M_PIf * 2.0f;
  const float theta = v * M_PIf; // theta == 0.0f is south pole, theta == M_PIf is north pole.

  const float sinTheta = sinf(theta);

  // All lights shine down the positive z-axis in this renderer.
  // Orient the 2D texture map so that the center (u, v) = (0.5, 0.5) lies exactly on the positive z-axis.
  // Means the seam from u == 1.0 -> 0.0 lies on the negative z-axis and the u range [0.0, 1.0]
  // goes clockwise on the xz-plane when looking from the positive y-axis.
  const float3 dir = make_float3( sinf(phi) * sinTheta,  // Starting on negative z-axis going around clockwise (to positive x-axis).
                                 -cosf(theta),           // From south pole to north pole.
                                 -cosf(phi) * sinTheta); // Starting on negative z-axis.
  
  // Now rotate that normalized object space direction into world space. 
  lightSample.direction = transformVector(light.ori, dir);

  lightSample.distance = RT_DEFAULT_MAX; // Environment light.
  
  // Get the emission from the spherical environment texture.
  const float3 emission = make_float3(tex2D<float4>(light.textureEmission, u, v));
  
  // For simplicity we pretend that we perfectly importance-sampled the actual texture-filtered environment map
  // and not the Gaussian-smoothed one used to actually generate the CDFs and uniform sampling in the texel.
  // (Note that this does not contain the light.emission which just modulates the texture.)
  lightSample.pdf = intensity(emission) * light.invIntegral;

  if (DENOMINATOR_EPSILON < lightSample.pdf)
  {
    lightSample.radiance_over_pdf = light.emission * emission / lightSample.pdf;
  }

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_point(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)
   
  // Get the world space position from the object to world matrix translation.
  const float3 position = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);

  lightSample.direction = position - prd->pos; // Sample direction from surface point to light sample position.
  
  const float distanceSquared = dot(lightSample.direction, lightSample.direction);

  if (DENOMINATOR_EPSILON < distanceSquared)
  {
    lightSample.distance   = sqrtf(distanceSquared);
    lightSample.direction *= 1.0f / lightSample.distance; // Normalized direction to light.

    // Hardcoded singular lights are defined in visible radiance directly, don't normalize by 0.25f * M_1_PIf.
    float3 emission = light.emission * (1.0f / distanceSquared); // Quadratic attenuation.

    // The emission texture is used as spherical projection around the point light similar to spherical environment lights.
    // By design all lights in this renderer shine down the light's local positive z-Axis, which is the "normal" direction for rect and mesh lights.
    if (light.textureEmission)
    {
      // Transform the direction from light to surface from world space into light object space.
      const float3 R = transformVector(light.oriInv, -lightSample.direction);

      // All lights shine down the positive z-axis in this renderer.
      // Means the spherical texture coordinate seam u == 0.0 == 1.0 is on the negative z-axis direction now.
      const float u = (atan2f(-R.x, R.z) + M_PIf) * 0.5f * M_1_PIf;
      const float v = acosf(-R.y) * M_1_PIf; // Texture is origin at lower left, v == 0.0f is south pole.

      // Modulate the base emission with the emission texture.
      emission *= make_float3(tex2D<float4>(light.textureEmission, u, v));
    }

    lightSample.radiance_over_pdf = emission; // pdf == 1.0f for singular light.

    // Indicate valid light sample (pdf != 0.0f).
    // This value is otherwise unused in a singular light. 
    // The PDF is a Dirac with infinite value for this case.
    lightSample.pdf = 1.0f;
  }

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_spot(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f;

  // Get the world space position from the world to object matrix translation.
  const float3 position = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);
  lightSample.direction = position - prd->pos; // Sample direction from surface point to light sample position.
  
  const float distanceSquared = dot(lightSample.direction, lightSample.direction);

  if (DENOMINATOR_EPSILON < distanceSquared)
  {
    lightSample.distance   = sqrtf(distanceSquared);
    lightSample.direction *= 1.0f / lightSample.distance; // Normalized direction to light.

    //const float3 normal = normalize(transformNormal(light.matrixInv, make_float3(0.0f, 0.0f, 1.0f)));
    const float3 normal = normalize(make_float3(light.matrixInv[2]));

    // Spot light is aligned to the local z-axis (the normal).
    const float cosTheta  = -dot(lightSample.direction, normal); // Negative because lightSample.direction is from surface to light.
    const float cosSpread = cosf(light.spotAngleHalf);           // Note that the spot light only supports hemispherical distributions.
   
    if (cosSpread <= cosTheta) // Is the lightSample.direction inside the spot light cone?
    {
      // Normalize the hemispherical distribution (half-angle M_PI_2f) to the cone angle (scale by factor: angleHalf / light.spotAngleHalf, range [0.0f, 1.0f]).
      const float cosHemi = cosf(M_PI_2f * acosf(cosTheta) / light.spotAngleHalf);

      // Hardcoded singular lights are defined in visible radiance directly, don't normalize.
      float3 emission = light.emission * (powf(cosHemi, light.spotExponent) / distanceSquared); // Quadratic attenuation.

      // The emission texture is used as projection scaled to the spherical cap inside the spot light cone.
      // By design all lights in this renderer shine down the light's local positive z-Axis, which is the "normal" direction for rect and mesh lights.
      if (light.textureEmission) 
      { 
        // Transform the direction from  light to surface from world space into light object space.
        const float3 R = transformVector(light.oriInv, -lightSample.direction);

        const float u = (acosf(R.x) - M_PI_2f) / light.spotAngleHalf * 0.5f + 0.5f;
        const float v = 0.5f - ((acosf(R.y) - M_PI_2f) / light.spotAngleHalf * 0.5f);

        // Modulate the base emission with the emission texture.
        emission *= make_float3(tex2D<float4>(light.textureEmission, u, v));
      }

      lightSample.radiance_over_pdf = emission; // pdf == 1.0f for singular light.

      // Indicate valid light sample (pdf != 0.0f).
      // This value is otherwise unused in a singular light. 
      // The PDF is a Dirac with infinite value for this case.
      lightSample.pdf = 1.0f;
    }
  }

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_ies(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)

  // Get the worls space position from the world to object matrix translation.
  const float3 position = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);
  lightSample.direction = position - prd->pos; // Sample direction from surface point to light sample position.
  
  const float distanceSquared = dot(lightSample.direction, lightSample.direction);

  if (DENOMINATOR_EPSILON < distanceSquared)
  {
    lightSample.distance   = sqrtf(distanceSquared);
    lightSample.direction *= 1.0f / lightSample.distance; // Normalized direction to light.

    // Hardcoded singular lights are defined in visible radiance directly, do not normalize.
    // This just returns the candela values (luminous power per solid angle).
    float3 emission = light.emission * (1.0f / distanceSquared);

    // The emission texture is used as spherical projection around the point light similar to spherical environment lights.
    // By design all lights in this renderer shine down the light's local positive z-Axis, which is the "normal" direction for rect and mesh lights.

    // Transform the direction from light to surface from world into light object space.
    const float3 R = transformVector(light.oriInv, -lightSample.direction);

    // All lights shine down the positive z-axis in this renderer.
    // Means the spherical texture coordinate seam u == 0.0 == 1.0 is on the negative z-axis direction now.
    const float u = (atan2f(-R.x, R.z) + M_PIf) * 0.5f * M_1_PIf;
    const float v = acosf(-R.y) * M_1_PIf; // Texture is origin at lower left, v == 0.0f is south pole.

    if (light.textureProfile)
    {
      // Modulate the base emission with the profile texture (single component float texture, candela)
      emission *= tex2D<float>(light.textureProfile, u, v);
    }

    if (light.textureEmission) // IES light profile can be modulated by emission color texture.
    {
      // Modulate the base emission with the emission texture.
      emission *= make_float3(tex2D<float4>(light.textureEmission, u, v));
    }

    lightSample.radiance_over_pdf = emission; // pdf == 1.0f for singular light.

    // Indicate valid light sample (pdf != 0.0f).
    // This value is otherwise unused in a singular light. 
    // The PDF is a Dirac with infinite value for this case.
    lightSample.pdf = 1.0f;
  }

  return lightSample;
}
