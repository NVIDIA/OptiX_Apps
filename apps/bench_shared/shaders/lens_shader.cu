#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"
#include "shader_common.h"

extern "C" __constant__ SystemData sysData;

// Note that all these lens shaders return the primary ray origin and direction in world space!

extern "C" __device__ void __direct_callable__pinhole(const float2 screen, const float2 pixel, const float2 sample, 
                                                      float3& origin, float3& direction)
{
  const float2 fragment = pixel + sample;                    // Jitter the sub-pixel location
  const float2 ndc      = (fragment / screen) * 2.0f - 1.0f; // Normalized device coordinates in range [-1, 1].

  const CameraDefinition camera = sysData.cameraDefinitions[0];

  origin    = camera.P;
  direction = normalize(camera.U * ndc.x +
                        camera.V * ndc.y +
                        camera.W);
}


extern "C" __device__ void __direct_callable__fisheye(const float2 screen, const float2 pixel, const float2 sample, 
                                                      float3& origin, float3& direction)
{
  const float2 fragment = pixel + sample; // x, y
  
  // Implement a fisheye projection with 180 degrees angle across the image diagonal (=> all pixels rendered, not a circular fisheye).
  const float2 center = screen * 0.5f;
  const float2 uv     = (fragment - center) / length(center); // uv components are in the range [0, 1]. Both 1 in the corners of the image!
  const float z       = cosf(length(uv) * 0.7071067812f * 0.5f * M_PIf); // Scale by 1.0f / sqrtf(2.0f) to get length into the range [0, 1]

  const CameraDefinition camera = sysData.cameraDefinitions[0];

  const float3 U = normalize(camera.U);
  const float3 V = normalize(camera.V);
  const float3 W = normalize(camera.W);

  origin    = camera.P;
  direction = normalize(uv.x * U + uv.y * V + z * W);
}


extern "C" __device__ void __direct_callable__sphere(const float2 screen, const float2 pixel, const float2 sample, 
                                                     float3& origin, float3& direction)
{
  const float2 uv = (pixel + sample) / screen; // "texture coordinates"

  // Convert the 2D index into a direction.
  const float phi   = uv.x * 2.0f * M_PIf;
  const float theta = uv.y * M_PIf;

  const float sinTheta = sinf(theta);

  const float3 v = make_float3(-sinf(phi) * sinTheta,
                               -cosf(theta),
                               -cosf(phi) * sinTheta);

  const CameraDefinition camera = sysData.cameraDefinitions[0];

  const float3 U = normalize(camera.U);
  const float3 V = normalize(camera.V);
  const float3 W = normalize(camera.W);

  origin    = camera.P;
  direction = normalize(v.x * U + v.y * V + v.z * W);
}
