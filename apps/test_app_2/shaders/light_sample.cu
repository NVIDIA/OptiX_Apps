#include "hip/hip_runtime.h"
 /* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"

#include "per_ray_data.h"
#include "random_number_generators.h"
#include "shader_common.h"
#include "transform.h"

extern "C" __constant__ SystemData sysData;


// Note that all light sampling routines return lightSample.direction and lightSample.distance in world space!

extern "C" __device__ LightSample __direct_callable__light_env_constant(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  const float2 sample = rng2(prd->seed);

  unitSquareToSphere(sample.x, sample.y, lightSample.direction, lightSample.pdf);

  // The emission is constant in all directions.
  // There is no transformation of the object space direction into world space necessary.

  lightSample.distance = RT_DEFAULT_MAX; // Environment light.
  
  lightSample.radiance_over_pdf = light.emission / lightSample.pdf;

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_env_sphere(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f;

  // Importance-sample the spherical environment light direction in object space.
  // FIXME The binary searches are generating a lot of memory traffic. Replace this with an alias-map lookup.
  const float2 sample = rng2(prd->seed);

  // Note that the marginal CDF is one bigger than the texture height. As index this is the 1.0f at the end of the CDF.
  const float* cdfV = reinterpret_cast<const float*>(light.cdfV);
  const unsigned int idxV = binarySearchCDF(cdfV, light.height, sample.y);

  const float* cdfU = reinterpret_cast<const float*>(light.cdfU);
  cdfU += (light.width + 1) * idxV; // Horizontal CDF is one bigger than the texture width!
  const unsigned int idxU = binarySearchCDF(cdfU, light.width, sample.x);

  // Continuous sampling of the CDF.
  const float cdfLowerU = cdfU[idxU];
  const float cdfUpperU = cdfU[idxU + 1];
  const float du = (sample.x - cdfLowerU) / (cdfUpperU - cdfLowerU);
  const float u = (float(idxU) + du) / float(light.width);

  const float cdfLowerV = cdfV[idxV];
  const float cdfUpperV = cdfV[idxV + 1];
  const float dv = (sample.y - cdfLowerV) / (cdfUpperV - cdfLowerV);
  const float v = (float(idxV) + dv) / float(light.height);

  // Light sample direction vector in object space polar coordinates.
  const float phi   = u * M_PIf * 2.0f;
  const float theta = v * M_PIf; // theta == 0.0f is south pole, theta == M_PIf is north pole.

  const float sinTheta = sinf(theta);

  // All lights shine down the positive z-axis in this renderer.
  // Orient the 2D texture map so that the center (u, v) = (0.5, 0.5) lies exactly on the positive z-axis.
  // Means the seam from u == 1.0 -> 0.0 lies on the negative z-axis and the u range [0.0, 1.0]
  // goes clockwise on the xz-plane when looking from the positive y-axis.
  const float3 dir = make_float3( sinf(phi) * sinTheta,  // Starting on negative z-axis going around clockwise (to positive x-axis).
                                 -cosf(theta),           // From south pole to north pole.
                                 -cosf(phi) * sinTheta); // Starting on negative z-axis.
  
  // Now rotate that normalized object space direction into world space. 
  lightSample.direction = transformVector(light.ori, dir);

  lightSample.distance = RT_DEFAULT_MAX; // Environment light.
  
  // Get the emission from the spherical environment texture.
  const float3 emission = make_float3(tex2D<float4>(light.textureEmission, u, v));
  
  // For simplicity we pretend that we perfectly importance-sampled the actual texture-filtered environment map
  // and not the Gaussian-smoothed one used to actually generate the CDFs and uniform sampling in the texel.
  // (Note that this does not contain the light.emission which just modulates the texture.)

  lightSample.pdf = intensity(emission) * light.invIntegral;

  if (DENOMINATOR_EPSILON < lightSample.pdf)
  {
    lightSample.radiance_over_pdf = light.emission * emission / lightSample.pdf;
  }

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_rect(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  // Default return, invalid light sample (backface, edge on, or too near to the surface)
  lightSample.radiance_over_pdf = make_float3(0.0f);
  lightSample.pdf               = 0.0f; 

  // Do the check for front or backface first before calculating any expensive sampling point.
  // Just use the center point to calculate the temporary direction.
  // This is sufficient because the rectangle light is planar.
  const float3 center = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);
  //const float3 normal = normalize(transformNormal(light.matrixInv, make_float3(0.0f, 0.0f, 1.0f)));
  const float3 normal = normalize(make_float3(light.matrixInv[2])); // The z-row of the inverse transpose matrix is the normal.

  // Check if the vector from center to surface is in the same hemisphere as the face normal.
  // Only need the sign, no need to normalize the temporary direction.
  if (0.0f < dot(prd->pos - center, normal))
  {
    float pdf = 1.0f; // Neutral factor in case there is no light.texture.

    // The diffuse EDF (per definition projected hemisphere) is 1/pi.
    // The PDF (non-projected hemisphere) ist cos/pi.
    // The radiance = EDF * radiant_exitance = 1/pi * radiant_exitance.
    float3 radiance = light.emission * M_1_PIf; 

    float2 sample = rng2(prd->seed);

    if (light.textureEmission)
    {
      // Importance-sample the rectangular texture in object space.
      // Note that the marginal CDF is one bigger than the texture height. As index this is the 1.0f at the end of the CDF.
      const float* cdfV = reinterpret_cast<const float*>(light.cdfV);
      const unsigned int idxV = binarySearchCDF(cdfV, light.height, sample.y);

      const float* cdfU = reinterpret_cast<const float*>(light.cdfU);
      cdfU += (light.width + 1) * idxV; // Horizontal CDF is one bigger than the texture width!
      const unsigned int idxU = binarySearchCDF(cdfU, light.width, sample.x);

      // Continuous sampling of the CDF.
      const float cdfLowerU = cdfU[idxU];
      const float cdfUpperU = cdfU[idxU + 1];
      const float du = (sample.x - cdfLowerU) / (cdfUpperU - cdfLowerU);
      sample.x = (float(idxU) + du) / float(light.width); // Texture coordinate and new sample.

      const float cdfLowerV = cdfV[idxV];
      const float cdfUpperV = cdfV[idxV + 1];
      const float dv = (sample.y - cdfLowerV) / (cdfUpperV - cdfLowerV);
      sample.y = (float(idxV) + dv) / float(light.height); // Texture coordinate and new sample.

      // Get the emission from the emission texture.
      const float3 emission = make_float3(tex2D<float4>(light.textureEmission, sample.x, sample.y));
      
      radiance *= emission; 

      // The pdf to have picked this emission on the texture.
      pdf = intensity(emission) * light.invIntegral; // This must be the emission from the texture only!
    }

    // Transform the default rectangle in the xy-range [-0.5, 0.5] into world space.
    //const float3 vecU   = transformVector(light.matrix, make_float3(1.0f, 0.0f, 0.0f));
    //const float3 vecV   = transformVector(light.matrix, make_float3(0.0f, 1.0f, 0.0f));
    //const float3 center = transformPoint(light.matrix, make_float3(0.0f));
    // Optimized versions.
    const float3 vecU   = make_float3(light.matrix[0].x, light.matrix[1].x, light.matrix[2].x);
    const float3 vecV   = make_float3(light.matrix[0].y, light.matrix[1].y, light.matrix[2].y);
    const float3 center = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);
  
    const float3 position = center + vecU * (sample.x - 0.5f) + vecV * (sample.y - 0.5f); // The light sample position in world coordinates.

    lightSample.direction = position - prd->pos; // Sample direction from surface point to light sample position.
    lightSample.distance  = length(lightSample.direction);
  
    if (DENOMINATOR_EPSILON < lightSample.distance)
    {
      lightSample.direction *= 1.0f / lightSample.distance; // Normalized direction to light.

      const float cosTheta = -dot(lightSample.direction, normal); // This must be greater than zero because of the initial frontface check.

      // Both PDFs multiplied! Latter is light area to solid angle (projected area) pdf. Assumes light.area != 0.0f.
      lightSample.pdf = pdf * lightSample.distance * lightSample.distance / (light.area * cosTheta);
      
      if (DENOMINATOR_EPSILON < lightSample.pdf)
      {
        lightSample.radiance_over_pdf = radiance / lightSample.pdf;
      }
    }
  }
  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_mesh(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;
 
  lightSample.pdf = 0.0f;

  const float3 sampleTriangle = rng3(prd->seed);

  // Uniformly sample the triangles over their surface area.
  // Note that zero-area triangles (e.g. at the poles of spheres) are automatically never sampled with this method!
  // The cdfU is one bigger than light.width.
  const float* cdfArea = reinterpret_cast<const float*>(light.cdfU);
  const unsigned int idxTriangle = binarySearchCDF(cdfArea, light.width, sampleTriangle.z);

  // Unit square to triangle via barycentric coordinates.
  const float su = sqrtf(sampleTriangle.x);
  // Barycentric coordinates.
  const float alpha = 1.0f - su;
  const float beta  = sampleTriangle.y * su;
  const float gamma = 1.0f - alpha - beta; 
  
  // This cast works because both unsigned int and uint3 have an alignment of 4 bytes.
  const uint3* indices = reinterpret_cast<uint3*>(light.indices);
  const uint3  tri     = indices[idxTriangle];
  
  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(light.attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  // Object space vertex attributes at the hit point.
  float3 po = attr0.vertex * alpha + attr1.vertex * beta + attr2.vertex * gamma;
  // Transform attributes into internal space == world space.
  po = transformPoint(light.matrix, po);

  const float3 texcoord = attr0.texcoord * alpha + attr1.texcoord * beta + attr2.texcoord * gamma;

  // Calculate the outgoing direction from light sample position to surface point.
  lightSample.direction = po - prd->pos;  // Sample direction from surface point to light sample position.
  lightSample.distance  = length(lightSample.direction);

  if (DENOMINATOR_EPSILON < lightSample.distance)
  {
    lightSample.direction *= 1.0f / lightSample.distance; // Normalized vector from light sample position to surface point.

    float3 normalGeo = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex); // Object space unnormalized geometry normal.
    normalGeo = normalize(transformNormal(light.matrixInv, normalGeo)); // World space geometry normal.

    // Negative cosine because lightSample.direction is from surface hit point to light sample position.
    const float cosTheta = -dot(lightSample.direction, normalGeo); 

    if (DENOMINATOR_EPSILON < cosTheta) // Only emit light on the front side of the light geometry.
    {
      // The diffuse EDF (per definition projected hemisphere) is 1/pi.
      // The PDF (non-projected hemisphere) ist cos/pi.
      // The radiance = EDF * radiant_exitance = 1/pi * radiant_exitance.
      float3 radiance = light.emission * M_1_PIf;

      if (light.textureEmission)
      {
        // Modulate the base emission with the emission texture. 
        // PERF Could sample first and compare with black.
        radiance *= make_float3(tex2D<float4>(light.textureEmission, texcoord.x, texcoord.y));
      }

      // Light area to solid angle (projected area) pdf. Assumes light.area != 0.0f.
      lightSample.pdf = lightSample.distance * lightSample.distance / (light.area * cosTheta); 

      if (DENOMINATOR_EPSILON < lightSample.pdf)
      {
        lightSample.radiance_over_pdf = radiance / lightSample.pdf;
      }
    }
  }
  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_point(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)

  // Get the world space position from the object to world matrix translation.
  const float3 position = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);

  lightSample.direction = position - prd->pos; // Sample direction from surface point to light sample position.
  
  const float distanceSquared = dot(lightSample.direction, lightSample.direction);

  if (DENOMINATOR_EPSILON < distanceSquared)
  {
    lightSample.distance   = sqrtf(distanceSquared);
    lightSample.direction *= 1.0f / lightSample.distance; // Normalized direction to light.

    // Hardcoded singular lights are defined in visible radiance directly, don't normalize by 0.25f * M_1_PIf.
    float3 emission = light.emission * (1.0f / distanceSquared); // Quadratic attenuation.

    // The emission texture is used as spherical projection around the point light similar to spherical environment lights.
    // By design all lights in this renderer shine down the light's local positive z-Axis, which is the "normal" direction for rect and mesh lights.
    if (light.textureEmission)
    {
      // Transform the direction from light to surface from world space into light object space.
      const float3 R = transformVector(light.oriInv, -lightSample.direction);

      // All lights shine down the positive z-axis in this renderer.
      // Means the spherical texture coordinate seam u == 0.0 == 1.0 is on the negative z-axis direction now.
      const float u = (atan2f(-R.x, R.z) + M_PIf) * 0.5f * M_1_PIf;
      const float v = acosf(-R.y) * M_1_PIf; // Texture is origin at lower left, v == 0.0f is south pole.

      // Modulate the base emission with the emission texture.
      emission *= make_float3(tex2D<float4>(light.textureEmission, u, v));
    }

    lightSample.radiance_over_pdf = emission; // pdf == 1.0f for singular light.

    // Indicate valid light sample (pdf != 0.0f).
    // This value is otherwise unused in a singular light. 
    // The PDF is a Dirac with infinite value for this case.
    lightSample.pdf = 1.0f;
  }

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_spot(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f;

  // Get the world space position from the world to object matrix translation.
  const float3 position = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);
  lightSample.direction = position - prd->pos; // Sample direction from surface point to light sample position.
  
  const float distanceSquared = dot(lightSample.direction, lightSample.direction);

  if (DENOMINATOR_EPSILON < distanceSquared)
  {
    lightSample.distance   = sqrtf(distanceSquared);
    lightSample.direction *= 1.0f / lightSample.distance; // Normalized direction to light.

    //const float3 normal = normalize(transformNormal(light.matrixInv, make_float3(0.0f, 0.0f, 1.0f)));
    const float3 normal = normalize(make_float3(light.matrixInv[2]));

    // Spot light is aligned to the local z-axis (the normal).
    const float cosTheta  = -dot(lightSample.direction, normal); // Negative because lightSample.direction is from surface to light.
    const float cosSpread = cosf(light.spotAngleHalf);           // Note that the spot light only supports hemispherical distributions.
   
    if (cosSpread <= cosTheta) // Is the lightSample.direction inside the spot light cone?
    {
      // Normalize the hemispherical distribution (half-angle M_PI_2f) to the cone angle (scale by factor: angleHalf / light.spotAngleHalf, range [0.0f, 1.0f]).
      const float cosHemi = cosf(M_PI_2f * acosf(cosTheta) / light.spotAngleHalf);

      // Hardcoded singular lights are defined in visible radiance directly, don't normalize.
      float3 emission = light.emission * (powf(cosHemi, light.spotExponent) / distanceSquared); // Quadratic attenuation.

      // The emission texture is used as projection scaled to the spherical cap inside the spot light cone.
      // By design all lights in this renderer shine down the light's local positive z-Axis, which is the "normal" direction for rect and mesh lights.
      if (light.textureEmission) 
      { 
        // Transform the direction from  light to surface from world space into light object space.
        const float3 R = transformVector(light.oriInv, -lightSample.direction);

        const float u = (acosf(R.x) - M_PI_2f) / light.spotAngleHalf * 0.5f + 0.5f;
        const float v = 0.5f - ((acosf(R.y) - M_PI_2f) / light.spotAngleHalf * 0.5f);

        // Modulate the base emission with the emission texture.
        emission *= make_float3(tex2D<float4>(light.textureEmission, u, v));
      }

      lightSample.radiance_over_pdf = emission; // pdf == 1.0f for singular light.

      // Indicate valid light sample (pdf != 0.0f).
      // This value is otherwise unused in a singular light. 
      // The PDF is a Dirac with infinite value for this case.
      lightSample.pdf = 1.0f;
    }
  }

  return lightSample;
}


extern "C" __device__ LightSample __direct_callable__light_ies(const LightDefinition& light, PerRayData* prd)
{
  LightSample lightSample;

  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)

  // Get the worls space position from the world to object matrix translation.
  const float3 position = make_float3(light.matrix[0].w, light.matrix[1].w, light.matrix[2].w);
  lightSample.direction = position - prd->pos; // Sample direction from surface point to light sample position.
  
  const float distanceSquared = dot(lightSample.direction, lightSample.direction);

  if (DENOMINATOR_EPSILON < distanceSquared)
  {
    lightSample.distance   = sqrtf(distanceSquared);
    lightSample.direction *= 1.0f / lightSample.distance; // Normalized direction to light.

    // Hardcoded singular lights are defined in visible radiance directly, do not normalize.
    // This just returns the candela values (luminous power per solid angle).
    float3 emission = light.emission * (1.0f / distanceSquared);

    // The emission texture is used as spherical projection around the point light similar to spherical environment lights.
    // By design all lights in this renderer shine down the light's local positive z-Axis, which is the "normal" direction for rect and mesh lights.

    // Transform the direction from light to surface from world into light object space.
    const float3 R = transformVector(light.oriInv, -lightSample.direction);

    // All lights shine down the positive z-axis in this renderer.
    // Means the spherical texture coordinate seam u == 0.0 == 1.0 is on the negative z-axis direction now.
    const float u = (atan2f(-R.x, R.z) + M_PIf) * 0.5f * M_1_PIf;
    const float v = acosf(-R.y) * M_1_PIf; // Texture is origin at lower left, v == 0.0f is south pole.

    if (light.textureProfile)
    {
      // Modulate the base emission with the profile texture (single component float texture, candela)
      emission *= tex2D<float>(light.textureProfile, u, v);
    }

    if (light.textureEmission) // IES light profile can be modulated by emission color texture.
    {
      // Modulate the base emission with the emission texture.
      emission *= make_float3(tex2D<float4>(light.textureEmission, u, v));
    }

    lightSample.radiance_over_pdf = emission; // pdf == 1.0f for singular light.

    // Indicate valid light sample (pdf != 0.0f).
    // This value is otherwise unused in a singular light. 
    // The PDF is a Dirac with infinite value for this case.
    lightSample.pdf = 1.0f;
  }

  return lightSample;
}
